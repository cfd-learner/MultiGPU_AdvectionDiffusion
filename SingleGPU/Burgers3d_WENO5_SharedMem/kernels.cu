#include "hip/hip_runtime.h"
//
//  kernels.cu
//  Burgers3d-GPU
//
//  Created by Manuel Diaz on 7/26/16.
//  Copyright © 2016 Manuel Diaz. All rights reserved.
//

extern "C" {
#include "Burgers.h"
}

/*******************************/
/* Define Textures & Constanst */
/*******************************/
__constant__ REAL d_kx;
__constant__ REAL d_ky;
__constant__ REAL d_kz;
texture<int2, 2, hipReadModeElementType> tex_u;

#define checkCuda(error) __checkCuda(error, __FILE__, __LINE__)

/*********************************************/
/* A method for checking error in CUDA calls */
/*********************************************/
inline void __checkCuda(hipError_t error, const char *file, const int line)
{
  #if defined(DISPL)
    if (error != hipSuccess)
    {
    printf("checkCuda error at %s:%i: %s\n", file, line, hipGetErrorString(hipGetLastError()));
    exit(-1);
    }
  #endif
  return;
}

/*****************/
/* FLUX FUNCTION */
/*****************/
__device__ REAL Flux(
  const REAL u){
  return 0.5*u*u;
}

/***********************/
/* WENO RECONSTRUCTION */
/***********************/

// *************************************************************************
// Input: v(i) = [v(i-2) v(i-1) v(i) v(i+1) v(i+2) v(i+3)];
// Output: res = df/dx;
//
// Based on:
// C.W. Shu's Lectures notes on: 'ENO and WENO schemes for Hyperbolic
// Conservation Laws'
//
// coded by Manuel Diaz, 02.10.2012, NTU Taiwan.
// *************************************************************************
//
// Domain cells (I{i}) reference:
//
//                |           |   u(i)    |           |
//                |  u(i-1)   |___________|           |
//                |___________|           |   u(i+1)  |
//                |           |           |___________|
//             ...|-----0-----|-----0-----|-----0-----|...
//                |    i-1    |     i     |    i+1    |
//                |-         +|-         +|-         +|
//              i-3/2       i-1/2       i+1/2       i+3/2
//
// ENO stencils (S{r}) reference:
//
//                           |___________S2__________|
//                           |                       |
//                   |___________S1__________|       |
//                   |                       |       |    using only f^{+}
//           |___________S0__________|       |       |
//         ..|---o---|---o---|---o---|---o---|---o---|...
//           | I{i-2}| I{i-1}|  I{i} | I{i+1}| I{i+2}|
//                                  -|
//                                 i+1/2
//
//                   |___________S0__________|
//                   |                       |
//                   |       |___________S1__________|    using only f^{-}
//                   |       |                       |
//                   |       |       |___________S2__________|
//                 ..|---o---|---o---|---o---|---o---|---o---|...
//                   | I{i-1}|  I{i} | I{i+1}| I{i+2}| I{i+3}|
//                                   |+
//                                 i+1/2
//
// WENO stencil: S{i} = [ I{i-2},...,I{i+3} ]
// *************************************************************************


__device__ REAL WENO5reconstruction(const REAL * __restrict__ u)
{
  REAL B0, B1, B2, a0, a1, a2, alphasum, dflux;
  REAL umm,um,uo,up,upp;

  // Split data for f_{i}^{+}
  umm=C1o2*(Flux(u[0]) + fabs(u[0])*u[0]);
  um =C1o2*(Flux(u[1]) + fabs(u[1])*u[1]);
  uo =C1o2*(Flux(u[2]) + fabs(u[2])*u[2]);
  up =C1o2*(Flux(u[3]) + fabs(u[3])*u[3]);
  upp=C1o2*(Flux(u[4]) + fabs(u[4])*u[4]);
  
  // Smooth Indicators (Beta factors)
  B0 = C13o12*(umm-2*um+uo )*(umm-2*um +uo ) + C1o4*(umm-4*um+3*uo)*(umm-4*um+3*uo);
  B1 = C13o12*(um -2*uo+up )*(um -2*uo +up ) + C1o4*(um-up)*(um-up);
  B2 = C13o12*(uo -2*up+upp)*(uo -2*up +upp) + C1o4*(3*uo-4*up+upp)*(3*uo-4*up+upp);
  
  // Alpha weights
  a0 = D0N/((EPS + B0)*(EPS + B0));
  a1 = D1N/((EPS + B1)*(EPS + B1));
  a2 = D2N/((EPS + B2)*(EPS + B2));
  alphasum = 1./(a0 + a1 + a2);
  
  // Numerical Flux at cell boundary, $v_{i+1/2}^{-}$;
  dflux =(a0*(2*umm- 7*um + 11*uo) +
          a1*( -um + 5*uo + 2*up) +
          a2*( 2*uo+ 5*up - upp ))*(C1o6*alphasum);

  // split data for f_{i}^{-}
  umm=C1o2*(Flux(u[1]) - fabs(u[1])*u[1]);
  um =C1o2*(Flux(u[2]) - fabs(u[2])*u[2]);
  uo =C1o2*(Flux(u[3]) - fabs(u[3])*u[3]);
  up =C1o2*(Flux(u[4]) - fabs(u[4])*u[4]);
  upp=C1o2*(Flux(u[5]) - fabs(u[5])*u[5]);

  // Smooth Indicators (Beta factors)
  B0 = C13o12*(umm-2*um+uo )*(umm-2*um +uo ) + C1o4*(umm-4*um+3*uo)*(umm-4*um+3*uo);
  B1 = C13o12*(um -2*uo+up )*(um -2*uo +up ) + C1o4*(um-up)*(um-up);
  B2 = C13o12*(uo -2*up+upp)*(uo -2*up +upp) + C1o4*(3*uo-4*up+upp)*(3*uo-4*up+upp);
  
  // Alpha weights
  a0 = D0P/((EPS + B0)*(EPS + B0));
  a1 = D1P/((EPS + B1)*(EPS + B1));
  a2 = D2P/((EPS + B2)*(EPS + B2));
  alphasum = 1./(a0 + a1 + a2);

  // Numerical Flux at cell boundary, $v_{i+1/2}^{+}$;
  dflux+=(a0*( -umm + 5*um + 2*uo ) +
          a1*( 2*um + 5*uo - up   ) +
          a2*(11*uo - 7*up + 2*upp))*(C1o6*alphasum);
  
  // Compute the numerical flux v_{i+1/2}
  return dflux;
}

__device__ REAL WENO5Zreconstruction(const REAL * __restrict__ u)
{
  REAL B0, B1, B2, a0, a1, a2, tau5, alphasum, dflux;
  REAL umm,um,uo,up,upp;

  // Split data for f_{i}^{+}
  umm=C1o2*(Flux(u[0]) + fabs(u[0])*u[0]);
  um =C1o2*(Flux(u[1]) + fabs(u[1])*u[1]);
  uo =C1o2*(Flux(u[2]) + fabs(u[2])*u[2]);
  up =C1o2*(Flux(u[3]) + fabs(u[3])*u[3]);
  upp=C1o2*(Flux(u[4]) + fabs(u[4])*u[4]);
  
  // Smooth Indicators (Beta factors)
  B0 = C13o12*(umm-2*um+uo )*(umm-2*um +uo ) + C1o4*(umm-4*um+3*uo)*(umm-4*um+3*uo);
  B1 = C13o12*(um -2*uo+up )*(um -2*uo +up ) + C1o4*(um-up)*(um-up);
  B2 = C13o12*(uo -2*up+upp)*(uo -2*up +upp) + C1o4*(3*uo-4*up+upp)*(3*uo-4*up+upp);
  // Alpha weights
  tau5 = fabs(B0-B2);
  a0 = D0N*(1.+tau5/(B0+EPS));
  a1 = D1N*(1.+tau5/(B1+EPS));
  a2 = D2N*(1.+tau5/(B2+EPS));
  alphasum = 1./(a0 + a1 + a2);
  
  // Numerical Flux at cell boundary, $v_{i+1/2}^{-}$;
  dflux =(a0*(2*umm- 7*um + 11*uo) +
          a1*( -um + 5*uo + 2*up) +
          a2*( 2*uo+ 5*up - upp ))*(C1o6*alphasum);

  // split data for f_{i}^{-}
  umm=C1o2*(Flux(u[1]) - fabs(u[1])*u[1]);
  um =C1o2*(Flux(u[2]) - fabs(u[2])*u[2]);
  uo =C1o2*(Flux(u[3]) - fabs(u[3])*u[3]);
  up =C1o2*(Flux(u[4]) - fabs(u[4])*u[4]);
  upp=C1o2*(Flux(u[5]) - fabs(u[5])*u[5]);

  // Smooth Indicators (Beta factors)
  B0 = C13o12*(umm-2*um+uo )*(umm-2*um +uo ) + C1o4*(umm-4*um+3*uo)*(umm-4*um+3*uo);
  B1 = C13o12*(um -2*uo+up )*(um -2*uo +up ) + C1o4*(um-up)*(um-up);
  B2 = C13o12*(uo -2*up+upp)*(uo -2*up +upp) + C1o4*(3*uo-4*up+upp)*(3*uo-4*up+upp);
  
  // Alpha weights
  tau5 = fabs(B0-B2);
  a0 = D0P*(1.+tau5/(B0+EPS));
  a1 = D1P*(1.+tau5/(B1+EPS));
  a2 = D2P*(1.+tau5/(B2+EPS));
  alphasum = 1./(a0 + a1 + a2);

  // Numerical Flux at cell boundary, $v_{i+1/2}^{+}$;
  dflux+=(a0*( -umm + 5*um + 2*uo ) +
          a1*( 2*um + 5*uo - up   ) +
          a2*(11*uo - 7*up + 2*upp))*(C1o6*alphasum);
  
  // Compute the numerical flux v_{i+1/2}
  return dflux;
}

/*****************/
/* Compute du/dx */ // <==== parallel strategy: compute serially by rows or by columns!
/*****************/
__global__ void Compute_dF(
  const REAL * __restrict__ u, 
  REAL * __restrict__ Lu, 
  const unsigned int pitch,
  const unsigned int nx, 
  const unsigned int ny, 
  const unsigned int nz, 
  const REAL dx)
{
  // Shared variables
  __shared__ REAL s_u[WIDTH][TILE+6]; // 3-wide halo
  __shared__ REAL sfu[WIDTH][TILE+1]; // faces = inner nodes + 1

  // Indexes
  unsigned int i,j,I,J,K,si,sj,o;
  
  // Global threads indexes
  I =    TILE    * blockIdx.x + threadIdx.x;
  J = blockDim.y * blockIdx.y + threadIdx.y;
  K = blockIdx.z;

  // Local thead indexes
  i = threadIdx.x;
  j = threadIdx.y;

  // Local share memory indexes
  si = threadIdx.x+3; // local i for shared memory access + halo offset
  sj = threadIdx.y;   // local j for shared memory access

  // Global index
  o = I+pitch*J+pitch*ny*K;

  if (I < nx){
    // Load data into shared memory
    s_u[sj][si]=u[o];

    // Load boundary values
    if ( (i<3) && (I<3) ){
      s_u[sj][si-3]=0.; // set Dirichlet BCs
    } else if (i < 3){
      s_u[sj][si-3]=u[o-3]; // get data from neighbour
    }

    // Load boundary values
    if ( (i>TILE-2) && (I>nx-2) ){
      s_u[sj][si+2]=0.; // set Dirichlet BCs
    } else if (i > TILE-2){
      s_u[sj][si+2]=u[o+2]; // get data from neighbour
    }
    __syncthreads();

    // Compute face fluxes
    sfu[j][i]=WENO5Zreconstruction(&s_u[sj][si-3]); // fp_{i+1/2}
    __syncthreads();
      
    // Compute Lq = (f_{i+1/2}-f_{i-1/2})/dx
    if ( i<TILE ){
      Lu[o] = -(sfu[j][i+1] - sfu[j][i])/dx;
    }
  } 
}

/*****************/
/* Compute du/dy */ // <==== parallel strategy: compute serialy by rows or by columns!
/*****************/
__global__ void Compute_dG(
  const REAL * __restrict__ v, 
  REAL * __restrict__ Lv, 
  const unsigned int pitch,
  const unsigned int nx, 
  const unsigned int ny, 
  const unsigned int nz, 
  const REAL dy)
{
  // Shared variables
  __shared__ REAL s_v[WIDTH][TILE+6]; // 3-wide halo
  __shared__ REAL sfv[WIDTH][TILE+1]; // faces = inner nodes + 1

  // Indexes
  unsigned int i,j,I,J,K,si,sj,o;
  
  // Global threads indexes
  I = blockDim.x * blockIdx.x + threadIdx.x;
  J =   TILE     * blockIdx.y + threadIdx.y;
  K = blockIdx.z;

  // Local thead indexes
  i = threadIdx.x;
  j = threadIdx.y;

  // Local share memory indexes
  si = threadIdx.x;   // local i for shared memory access
  sj = threadIdx.y+3; // local j for shared memory access + halo offset

  // Global index
  o = I+pitch*J+pitch*ny*K;

  if (J < ny){
    // Load data into shared memory
    s_v[si][sj]=v[o];

    // Load boundary values
    if ( (j<3) && (J<3) ){
      s_v[si][sj-3]=0.; // set Dirichlet BCs
    } else if (j < 3){
      s_v[si][sj-3]=v[o-3*pitch]; // get data from neighbour
    }

    // Load boundary values
    if ( (j>TILE-2) && (J>ny-2) ){
      s_v[si][sj+2]=0.; // set Dirichlet BCs
    } else if (j > TILE-2){
      s_v[si][sj+2]=v[o+2*pitch]; // get data from neighbour
    }
    __syncthreads();

    // Compute face fluxes
    sfv[i][j]=WENO5Zreconstruction(&s_v[si][sj-3]); // fp_{i+1/2}
    __syncthreads();
      
    // Compute Lq = (f_{i+1/2}-f_{i-1/2})/dx
    if ( j<TILE ){
      Lv[o] -= (sfv[i][j+1] - sfv[i][j])/dy;
    }
  }
}

/*****************/
/* Compute du/dz */ // <==== parallel strategy: compute serialy by rows or by columns!
/*****************/
__global__ void Compute_dH(
  const REAL * __restrict__ w, 
  REAL * __restrict__ Lw,
  const unsigned int pitch,
  const unsigned int nx, 
  const unsigned int ny, 
  const unsigned int nz, 
  const REAL dz)
{
  // Shared variables
  __shared__ REAL s_w[WIDTH][TILE+6]; // 3-wide halo
  __shared__ REAL sfw[WIDTH][TILE+1]; // faces = inner nodes + 1

  // Indexes
  unsigned int i,k,I,J,K,si,sk,o;

  // Global threads indexes
  I = blockDim.x * blockIdx.x + threadIdx.x;
  K =   TILE     * blockIdx.y + threadIdx.y;
  J = blockIdx.z;

  // Local thead indexes
  i = threadIdx.x;
  k = threadIdx.y;

  // Local share memory indexes
  si = threadIdx.x;   // local i for shared memory access
  sk = threadIdx.y+3; // local j for shared memory access + halo offset

  // Global index
  o = I+pitch*J+pitch*ny*K;

  if (K < nz){
    // Load data into shared memory
    s_w[si][sk]=w[o];

    // Load boundary values
    if ( (k<3) && (K<3) ){
      s_w[si][sk-3]=0.; // set Dirichlet BCs
    } else if (k < 3){
      s_w[si][sk-3]=w[o-3*pitch*ny]; // get data from neighbour
    }

    // Load boundary values
    if ( (k>TILE-2) && (K>nz-2) ){
      s_w[si][sk+2]=0.; // set Dirichlet BCs
    } else if (k > TILE-2){
      s_w[si][sk+2]=w[o+2*pitch*ny]; // get data from neighbour
    }
    __syncthreads();

    // Compute face fluxes
    sfw[i][k]=WENO5Zreconstruction(&s_w[si][sk-3]); // fp_{i+1/2}
    __syncthreads();
      
    // Compute Lq = (f_{i+1/2}-f_{i-1/2})/dz
    if ( k<TILE ){
      Lw[o] -= (sfw[i][k+1] - sfw[i][k])/dz;
    }
  }
}

/********************/
/* Laplace Operator */
/********************/
__global__ void Compute_Laplace(
  const REAL * __restrict__ u, 
  REAL * __restrict__ Lu,  
  const unsigned int px, // pitch in the x-direction
  const unsigned int nx, 
  const unsigned int ny, 
  const unsigned int nz)
{
  REAL above2;
  REAL above;
  REAL center;
  REAL below;
  REAL below2;
  unsigned int i, j, k, o, xy, px2, xy2;
  xy = px*ny; px2 = 2*px; xy2 = 2*xy; 

    i = threadIdx.x + blockIdx.x * blockDim.x;
    j = threadIdx.y + blockIdx.y * blockDim.y;

    // For initial slice
    k=3; o=i+px*j+xy*k;

    if (i>2 && i<nx-3 && j>2 && j<ny-3)
    {
      below2=u[o-xy2]; below=u[o-xy]; center=u[o]; above=u[o+xy]; above2=u[o+xy2];

      Lu[o]+= d_kx * (- u[o-2] +16*u[o-1] - 30*center + 16*u[o+1] - u[o+2] ) +
              d_ky * (-u[o-px2]+16*u[o-px]- 30*center + 16*u[o+px]- u[o+px2])+ 
              d_kz * (- below2 +16* below - 30*center + 16* above - above2 );

      // For the rest of the slide
      for(k = 4; k < nz-3; k++)
      {
        o=o+xy; below2=below; below=center; center=above; above=above2; above2=u[o+xy2];

        Lu[o]+= d_kx * (- u[o-2] +16*u[o-1] - 30*center + 16*u[o+1] - u[o+2] ) +
                d_ky * (-u[o-px2]+16*u[o-px]- 30*center + 16*u[o+px]- u[o+px2])+ 
                d_kz * (- below2 +16* below - 30*center + 16* above - above2 );
      }
    }
    // else : do nothing!
}

/**************************/
/* Async Laplace Operator */
/**************************/
__global__ void Compute_Laplace_Async(
  const REAL * __restrict__ u, 
  REAL * __restrict__ Lu, 
  unsigned int px, // pitch in the x-direction
  unsigned int Nx, 
  unsigned int Ny, 
  unsigned int _Nz, 
  unsigned int kstart, 
  unsigned int kstop, 
  unsigned int loop_z)
{
  register REAL above2;
  register REAL above;
  register REAL center;
  register REAL below;
  register REAL below2;
  unsigned int i, j, k, o, z, XY, px2, XY2;

  i = blockIdx.x * blockDim.x + threadIdx.x;
  j = blockIdx.y * blockDim.y + threadIdx.y;
  k = blockIdx.z * loop_z;

  k = MAX(kstart,k);

  XY=px*Ny; px2=px+px; XY2=XY+XY; o=i+px*j+XY*k;

  if (i>2 && i<Nx-3 && j>2 && j<Ny-3)
  {
    below2=u[o-XY2]; below=u[o-XY]; center=u[o]; above=u[o+XY]; above2=u[o+XY2];

    Lu[o]+= d_kx*(- u[o-2] +16* u[o-1]-30*center+16*u[o+1] - u[o+2] ) +
            d_ky*(-u[o-px2]+16*u[o-px]-30*center+16*u[o+px]-u[o+px2]) +
            d_kz*(- below2 +16* below -30*center+16* above - above2 );
      

    for(z = 1; z < loop_z; z++)
    {
      k += 1;

      if (k < MIN(kstop,_Nz+1))
      {
        o=o+XY; below2=below; below=center; center=above; above=above2; above2=u[o+XY2];

        Lu[o]+= d_kx*(- u[o-2] +16*u[o-1] -30*center+16*u[o+1] - u[o+2] ) + 
                d_ky*(-u[o-px2]+16*u[o-px]-30*center+16*u[o+px]-u[o+px2]) + 
                d_kz*(- below2 +16* below -30*center+16* above - above2 );
      }
    }
  }
  // else : do nothing!
}

/***********************/
/* Runge Kutta Methods */  // <==== this is perfectly parallel!
/***********************/
__global__ void Compute_RK( 
  REAL * __restrict__ u, 
  const REAL * __restrict__ uo, 
  const REAL * __restrict__ Lu, 
  const unsigned int step, 
  const unsigned int pitch, 
  const unsigned int nx, 
  const unsigned int ny, 
  const unsigned int nz, 
  const REAL dt)
{
  // local threads indexes
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int k = blockDim.z * blockIdx.z + threadIdx.z;
    
  // compute single index
  unsigned int o=i+pitch*j+pitch*ny*k;

  // Compute Runge-Kutta step, update only internal cells
  if (i>2 && i<nx-3 && j>2 && j<ny-3 && k>2 && k<nz-3)
  {
    switch (step) {
      case 1: // step 1
          u[o] = uo[o]+dt*(Lu[o]); break;
      case 2: // step 2
          u[o] = 0.75*uo[o]+0.25*(u[o]+dt*(Lu[o])); break;
      case 3: // step 3
          u[o] = (uo[o]+2*(u[o]+dt*(Lu[o])))/3; break;
    }
  }
  // else : do nothing!
}

/*********************/
/* Function Wrappers */
/*********************/
extern "C" void CopyToConstantMemory(const REAL kx, const REAL ky, const REAL kz)
{
  checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_kx), &kx, sizeof(REAL), 0, hipMemcpyHostToDevice));
  checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_ky), &ky, sizeof(REAL), 0, hipMemcpyHostToDevice));
  checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_kz), &kz, sizeof(REAL), 0, hipMemcpyHostToDevice));
}

extern "C" void Call_Adv_x(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream, 
  unsigned int pitch, unsigned int nx, unsigned int ny, unsigned int nz, REAL dx, REAL *u, REAL *Lu)
{
  Compute_dF<<<numBlocks,threadsPerBlock,0,aStream>>>(u,Lu,pitch,nx,ny,nz,dx);
}

extern "C" void Call_Adv_y(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream, 
  unsigned int pitch, unsigned int nx, unsigned int ny, unsigned int nz, REAL dy, REAL *u, REAL *Lu)
{
  Compute_dG<<<numBlocks,threadsPerBlock,0,aStream>>>(u,Lu,pitch,nx,ny,nz,dy);
}

extern "C" void Call_Adv_z(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream, 
  unsigned int pitch, unsigned int nx, unsigned int ny, unsigned int nz, REAL dz, REAL *u, REAL *Lu)
{
  Compute_dH<<<numBlocks,threadsPerBlock,0,aStream>>>(u,Lu,pitch,nx,ny,nz,dz);
}

extern "C" void Call_Diff_(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream,
  unsigned int pitch, unsigned int nx, unsigned int ny, unsigned int nz, REAL *u, REAL *Lu)
{
  // Compute_Laplace<<<numBlocks,threadsPerBlock,0,aStream>>>(u,Lu,pitch,nx,ny,nz);
  Compute_Laplace_Async<<<numBlocks,threadsPerBlock,0,aStream>>>(u,Lu,pitch,nx,ny,nz,3,nz-2,LOOP);
}

extern "C" void Call_sspRK(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream, 
  unsigned int pitch, unsigned int nx, unsigned int ny, unsigned int nz, unsigned int step, REAL dt, REAL *u, REAL *uo, REAL *Lu)
{
  Compute_RK<<<numBlocks,threadsPerBlock,0,aStream>>>(u,uo,Lu,step,pitch,nx,ny,nz,dt);
}
