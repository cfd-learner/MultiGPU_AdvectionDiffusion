#include "hip/hip_runtime.h"
//
//  kernels.cu
//  diffusion3d-GPU
//
//  Created by Manuel Diaz on 7/26/16.
//  Copyright © 2016 Manuel Diaz. All rights reserved.
//

extern "C" {
#include "acoustics3d.h"
}

#define checkCuda(error) __checkCuda(error, __FILE__, __LINE__)

__constant__ REAL d_kx;
__constant__ REAL d_ky;
__constant__ REAL d_kz;

/*********************************************/
/* A method for checking error in CUDA calls */
/*********************************************/
inline void __checkCuda(hipError_t error, const char *file, const int line)
{
  #if defined(DISPL)
    if (error != hipSuccess)
    {
    printf("checkCuda error at %s:%i: %s\n", file, line, hipGetErrorString(hipGetLastError()));
    exit(-1);
    }
  #endif
  return;
}

/********************/
/* Laplace Operator */
/********************/
__global__ void Compute_Laplace3d_O2(
  REAL * __restrict__ u, 
  REAL * __restrict__ Lu,  
  const unsigned int nx, 
  const unsigned int ny, 
  const unsigned int nz)
{
  unsigned int i, j, k, o, n, s, e, w, t, b, xy;
  xy = nx*ny;

    i = threadIdx.x + blockIdx.x * blockDim.x;
    j = threadIdx.y + blockIdx.y * blockDim.y;
    k = threadIdx.z + blockIdx.z * blockDim.z;

    o = i+(nx*j)+(xy*k); // node( j,i,k )      n  b
    n = o+nx;            // node(j+1,i,k)      | /
    s = o-nx;            // node(j-1,i,k)      |/ 
    e = o+1;             // node(j,i+1,k)  w---o---e
    w = o-1;             // node(j,i-1,k)     /|
    t = o+xy;            // node(j,i,k+1)    / |
    b = o-xy;            // node(j,i,k-1)   t  s
    
    if (i>0 && i<nx-1 && j>0 && j<ny-1 && k>0 && k<nz-1)
      Lu[o] = 12*d_kx*(u[e]-2*u[o]+u[w])+
              12*d_ky*(u[n]-2*u[o]+u[s])+
              12*d_kz*(u[t]-2*u[o]+u[b]);
    // else : do nothing!
}

__global__ void Compute_Laplace3d(
  REAL * __restrict__ u, 
  REAL * __restrict__ Lu,  
  const unsigned int nx, 
  const unsigned int ny, 
  const unsigned int nz)
{
  unsigned int i, j, k, o, n, s, e, w, t, b, nn, ss, ee, ww, tt, bb, xy;
  xy = nx*ny;

    i = threadIdx.x + blockIdx.x * blockDim.x;
    j = threadIdx.y + blockIdx.y * blockDim.y;
    k = threadIdx.z + blockIdx.z * blockDim.z;

    o = i+(nx*j)+(xy*k); // node( j,i,k )        nn    bb
    nn= o+nx+nx;         // node(j+2,i,k)        |    /
    n = o+nx;            // node(j+1,i,k)        |   /
    s = o-nx;            // node(j-1,i,k)        n  b
    ss= o-nx-nx;         // node(j-2,i,k)        | /
    ee= o+2;             // node(j,i+2,k)        |/
    e = o+1;             // node(j,i+1,k) ww--w--o--e--ee
    w = o-1;             // node(j,i-1,k)       /|
    ww= o-2;             // node(j,i-2,k)      / |
    tt= o+xy+xy;         // node(j,i,k+2)     t  s
    t = o+xy;            // node(j,i,k+1)    /   |
    b = o-xy;            // node(j,i,k-1)   /    |
    bb= o-xy-xy;         // node(j,i,k-2)  tt    ss

    if (i>1 && i<nx-2 && j>1 && j<ny-2 && k>1 && k<nz-2)
      Lu[o] = d_kx*(-u[ee]+16*u[e]-30*u[o]+16*u[w]-u[ww])+ 
              d_ky*(-u[nn]+16*u[n]-30*u[o]+16*u[s]-u[ss])+ 
              d_kz*(-u[tt]+16*u[t]-30*u[o]+16*u[b]-u[bb]);
    // else : do nothing!
}

/***********************/
/* Runge Kutta Methods */  // <==== this is perfectly parallel!
/***********************/
__global__ void Compute_RK( 
  REAL * __restrict__ u, 
  const REAL * __restrict__ uo, 
  const REAL * __restrict__ Lu, 
  const unsigned int step, 
  const unsigned int nx, 
  const unsigned int ny, 
  const unsigned int nz, 
  const REAL dt)
{
  // Using (o) = [i+nx*j+nx*ny*k] index
  unsigned int i, j, k, o, xy; 
  xy=nx*ny;

  // Compute Runge-Kutta step, local threads indexes
  i = blockDim.x * blockIdx.x + threadIdx.x;
  j = blockDim.y * blockIdx.y + threadIdx.y;
  k = blockDim.z * blockIdx.z + threadIdx.z;
    
  // compute single index
  o=i+nx*j+xy*k;

  // update only internal cells
  if (i>1 && i<nx-2 && j>1 && j<ny-2 && k>1 && k<nz-2)
  {
    switch (step) {
      case 1: // step 1
          u[o] = uo[o]+dt*(Lu[o]);
          break;
      case 2: // step 2
          u[o] = 0.75*uo[o]+0.25*(u[o]+dt*(Lu[o]));
          break;
      case 3: // step 3
          u[o] = (uo[o]+2*(u[o]+dt*(Lu[o])))/3;
          break;
    }
  }
  // else : do nothing!
}

/*********************/
/* Function Wrappers */
/*********************/
extern "C" void CopyToConstantMemory(const REAL kx, const REAL ky, const REAL kz)
{
  checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_kx), &kx, sizeof(REAL), 0, hipMemcpyHostToDevice));
  checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_ky), &ky, sizeof(REAL), 0, hipMemcpyHostToDevice));
  checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_kz), &kz, sizeof(REAL), 0, hipMemcpyHostToDevice));
}

extern "C" void Call_Lu3d(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream, 
  unsigned int nx, unsigned int ny, unsigned int nz, REAL *u, REAL *Lu)
{
  Compute_Laplace3d<<<numBlocks,threadsPerBlock,0,aStream>>>(u,Lu,nx,ny,nz);
}

extern "C" void Call_RK3d(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream, 
  unsigned int step, unsigned int nx, unsigned int ny, unsigned int nz, const REAL dt, REAL *u, REAL *uo, REAL *Lu)
{
  Compute_RK<<<numBlocks,threadsPerBlock,0,aStream>>>(u,uo,Lu,step,nx,ny,nz,dt);
}
