#include "hip/hip_runtime.h"
//
//  kernels.cu
//  diffusion3d-GPU
//
//  Created by Manuel Diaz on 7/26/16.
//  Copyright © 2016 Manuel Diaz. All rights reserved.
//

extern "C" {
#include "acoustics3d.h"
}

#define checkCuda(error) __checkCuda(error, __FILE__, __LINE__)

__constant__ REAL d_kx;
__constant__ REAL d_ky;
__constant__ REAL d_kz;

/*********************************************/
/* A method for checking error in CUDA calls */
/*********************************************/
inline void __checkCuda(hipError_t error, const char *file, const int line)
{
  #if defined(DISPL)
    if (error != hipSuccess)
    {
    printf("checkCuda error at %s:%i: %s\n", file, line, hipGetErrorString(hipGetLastError()));
    exit(-1);
    }
  #endif
  return;
}

/********************/
/* Laplace Operator */
/********************/
__global__ void Compute_Laplace3d_Async(
  const REAL * __restrict__ u, 
  REAL * __restrict__ Lu, 
  const unsigned int px, // allocation pitch
  const unsigned int Nx, 
  const unsigned int Ny, 
  const unsigned int _Nz, 
  const unsigned int kstart, 
  const unsigned int kstop, 
  const unsigned int loop_z)
{
  register REAL above2;
  register REAL above;
  register REAL center;
  register REAL below;
  register REAL below2;
  unsigned int z, XY, Nx2, XY2;

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * loop_z;

  k = MAX(kstart,k);

  XY=px*Ny; Nx2=px+px; XY2=XY+XY; 

  int o=i+px*j+XY*k;

  if (i>2 && i<Nx-3 && j>2 && j<Ny-3)
  {
    below2=u[o-XY2]; below=u[o-XY]; center=u[o]; above=u[o+XY]; above2=u[o+XY2];

    Lu[o] = d_kx * (- u[o-2] + 16*u[o-1] - 30*center + 16*u[o+1] - u[o+2]) + 
            d_ky * (-u[o-Nx2]+ 16*u[o-px]- 30*center + 16*u[o+px]- u[o+Nx2]) +
            d_kz * (- below2 + 16*below  - 30*center + 16* above - above2 );

    for(z = 1; z < loop_z; z++)
    {
      k += 1;

      if (k < MIN(kstop,_Nz+1))
      {
        o=o+XY; below2=below; below=center; center=above; above=above2; above2=u[o+XY2];

        Lu[o] = d_kx * (- u[o-2] + 16*u[o-1] - 30*center + 16*u[o+1] - u[o+2]) +
                d_ky * (-u[o-Nx2]+ 16*u[o-px]- 30*center + 16*u[o+px]- u[o+Nx2]) +
                d_kz * (- below2 + 16* below - 30*center + 16* above - above2 );
      }
    }
  }
  // else : do nothing!
}

/***********************/
/* Runge Kutta Methods */  // <==== this is perfectly parallel!
/***********************/
__global__ void Compute_RK( 
  REAL * __restrict__ u, 
  const REAL * __restrict__ uo, 
  const REAL * __restrict__ Lu, 
  const unsigned int step, 
  const unsigned int pitch,
  const unsigned int nx, 
  const unsigned int ny, 
  const unsigned int nz, 
  const REAL dt)
{
  // Compute Runge-Kutta step, local threads indexes
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int k = blockDim.z * blockIdx.z + threadIdx.z;
    
  // compute single index
  int o=i+pitch*j+pitch*ny*k;

  // update only internal cells
  if (i>1 && i<nx-2 && j>1 && j<ny-2 && k>1 && k<nz-2)
  {
    switch (step) {
      case 1: // step 1
          u[o] = uo[o]+dt*(Lu[o]);
          break;
      case 2: // step 2
          u[o] = 0.75*uo[o]+0.25*(u[o]+dt*(Lu[o]));
          break;
      case 3: // step 3
          u[o] = (uo[o]+2*(u[o]+dt*(Lu[o])))/3;
          break;
    }
  }
  // else : do nothing!
}

__global__ void Compute_RK_Async( 
  REAL * __restrict__ q, 
  const REAL * __restrict__ qo, 
  const REAL * __restrict__ Lq, 
  const unsigned int step,
  const unsigned int pitch, 
  const unsigned int Nx, 
  const unsigned int Ny, 
  const unsigned int _Nz, 
  const unsigned int kstart, 
  const unsigned int kstop, 
  const unsigned int loop_z, 
  const REAL dt)
{
  int z, XY = pitch*Ny;
  // local threads indexes
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int k = blockIdx.z * loop_z;

  k = MAX(kstart,k);

  // Single index
  int o=i+pitch*j+XY*k;

  // Compute Runge-Kutta step only on internal cells
  if (i>1 && i<Nx-2 && j>1 && j<Ny-2)
  {
    for(z = 0; z < loop_z; z++)
    {
      if (k < MIN(kstop,_Nz-2)) 
      {
        switch (step) {
          case 1: // step 1
            q[o] = qo[o]+dt*(Lq[o]); break;
          case 2: // step 2
            q[o] = 0.75*qo[o]+0.25*(q[o]+dt*(Lq[o])); break;
          case 3: // step 3
            q[o] = (qo[o]+2*(q[o]+dt*(Lq[o])))/3; break;
        }
        o += XY;
      }
      k += 1;
    }
  }
}

/*********************/
/* Function Wrappers */
/*********************/
extern "C" void CopyToConstantMemory(REAL kx, REAL ky, REAL kz)
{
  checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_kx), &kx, sizeof(REAL), 0, hipMemcpyHostToDevice));
  checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_ky), &ky, sizeof(REAL), 0, hipMemcpyHostToDevice));
  checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_kz), &kz, sizeof(REAL), 0, hipMemcpyHostToDevice));
}

extern "C" void Call_Lu3d(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream, 
  unsigned int pitch, unsigned int nx, unsigned int ny, unsigned int nz, REAL *q, REAL *Lq)
{
  Compute_Laplace3d_Async<<<numBlocks,threadsPerBlock,0,aStream>>>(q,Lq,pitch,nx,ny,nz,3,nz-2,k_loop);
}

extern "C" void Call_RK3d(dim3 numBlocks, dim3 threadsPerBlock, hipStream_t aStream, 
  unsigned int step, unsigned int pitch, unsigned int nx, unsigned int ny, unsigned int nz, REAL dt, REAL *q, REAL *qo, REAL *Lq)
{
  Compute_RK<<<numBlocks,threadsPerBlock,0,aStream>>>(q,qo,Lq,step,pitch,nx,ny,nz,dt);
  // Compute_RK_Async<<<numBlocks,threadsPerBlock,0,aStream>>>(q,qo,Lq,step,pitch,nx,ny,nz,3,nz-2,k_loop,dt);
}
